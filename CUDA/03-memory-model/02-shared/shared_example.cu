
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cassert>


#define BLOCKSIZE 512

__global__ void ComputeThreeSum(int n, int* input, int* result) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int local_tid = threadIdx.x;
    __shared__ int s_data[BLOCKSIZE + 2]; // unique for every block!
    if (local_tid == 0 && tid > 0) {
        s_data[0] = input[tid - 1];
    } else if (local_tid == blockDim.x - 1 && tid + 1 < n) {
        s_data[BLOCKSIZE + 1] = input[tid + 1];
    }
    s_data[local_tid + 1] = input[tid]; // copy data to shared memory
    
    __syncthreads();

    result[tid] = s_data[local_tid] + s_data[local_tid + 1] + s_data[local_tid + 2]; 

}


int main() {
    int N = 1 << 28;

    int* h_array = new int[N];
    int* h_diff = new int[N];
    for (int i = 0; i < N; ++i) {
        h_array[i] = 1;
    }
    
    int* d_array;
    int* d_diff;
    unsigned int size = N * sizeof(int);
    hipMalloc(&d_array, size);
    hipMalloc(&d_diff, size);

    hipMemcpy(d_array, h_array, size, hipMemcpyHostToDevice);
    
    int num_blocks = (N + BLOCKSIZE - 1) / BLOCKSIZE;

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    ComputeThreeSum<<<num_blocks, BLOCKSIZE>>>(N, d_array, d_diff);
    hipEventRecord(stop);


    hipMemcpy(h_diff, d_diff, size, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds;


    hipEventElapsedTime(&milliseconds, start, stop);

    for (int i = 1; i < N - 1; ++i) {
        if (h_diff[i] != 3) {
            std::cout << i << " " << h_diff[i] << std::endl;    
        }
        assert(h_diff[i] == 3);
    }

    std::cout << milliseconds << " elapsed" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_array);
    hipFree(d_diff);
    delete[] h_array;
    delete[] h_diff;

}
