
#include <hip/hip_runtime.h>
#include <iostream>

#define BLOCKSIZE 1024

__global__ void StupidSumArray(int* array, int* result) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    result[tid] = 0;
    for (int i = tid * 1024; i < (tid + 1) * 1024; ++i) {
        result[tid] += array[i];
    }
}


int main() {
    int N = 1 << 20;
    int *h_x = new int[N];

    for (int i = 0; i < N; ++i) {
        h_x[i] = 1;
    }
    int *d_x;
    int size = sizeof(int) * N;
    hipMalloc(&d_x, size);

    int* h_result = new int[1024];
    for (int i = 0; i < BLOCKSIZE; ++i) {
        h_result[i] = 0;
    }
    int *d_result;
    hipMalloc(&d_result, sizeof(int) * 1024); 

    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_result, h_result, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    StupidSumArray<<<1, 1024>>>(d_x, d_result);

    hipEventRecord(stop);

    hipMemcpy(h_result, d_result, sizeof(int) * 1024, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    for (int i = 0; i < 1024; ++i) {
        std::cout << i << " " << h_result[i] << std::endl;
    }

    std::cout << ms << std::endl;
    hipFree(d_x);
    hipFree(d_result);
    delete[] h_result;
    delete[] h_x;

}
