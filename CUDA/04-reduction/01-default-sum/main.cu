
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void Reduce(int* in_data, int* out_data) {
    extern __shared__ int shared_data[];

    unsigned int tid = threadIdx.x;
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    shared_data[tid] = in_data[index];
    __syncthreads();
    
    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) {
            shared_data[tid] += shared_data[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        out_data[blockIdx.x] = shared_data[0];
    }
}


int main(int argc, char** argv) {
    const int block_size = 1024;
    // __shared__ int shared_data[];

    const int array_size = 1 << 28;
    int* h_array = new int[array_size];
    for (int i = 0; i < array_size; ++i) {
        h_array[i] = 1;
    }

    int* d_array;
    hipMalloc(&d_array, sizeof(int) * array_size);

    hipMemcpy(d_array, h_array, sizeof(int) * array_size, hipMemcpyHostToDevice);

    int num_blocks = array_size / block_size;

    int* d_blocksum;
    hipMalloc(&d_blocksum, sizeof(int) * num_blocks);
    int* h_blocksum = new int[num_blocks];

    hipEvent_t start;
    hipEvent_t stop;

    // Creating event
    hipEventCreate(&start);
    hipEventCreate(&stop);


    hipEventRecord(start);

    Reduce<<<num_blocks, block_size, sizeof(int) * block_size>>>(d_array, d_blocksum);

    hipEventRecord(stop);

    hipMemcpy(h_blocksum, d_blocksum, sizeof(int) * num_blocks, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);

    float milliseconds = 0;

    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << milliseconds << " elapsed" << std::endl;
    
    int sum = 0;
    for (int i = 0; i < num_blocks; ++i) {
        sum += h_blocksum[i];
    }

    std::cout << sum << std::endl;

    hipFree(d_blocksum);
    hipFree(d_array);
    delete[] h_array;
    delete[] h_blocksum;

}
