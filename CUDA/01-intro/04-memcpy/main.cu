
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cstdio>

#define cudaErrchk(ans) { GPUAssert((ans), __FILE__, __LINE__); }
inline void GPUAssert(hipError_t code, const char *file, int line, bool abort=true){
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__
void add(int n, float* x, float* y) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < n) {
		y[index] = x[index] + y[index];
	}
}


int main() {
	int N = 1 << 28;
	size_t size = N * sizeof(float);
	float *h_x = (float*)malloc(size);
	float *h_y = (float*)malloc(size);

	float *d_x, *d_y;

	hipMalloc(&d_x, size);
	hipMalloc(&d_y, size);


	for (int i = 0; i < N; ++i) {
		h_x[i] = 1.0f;
		h_y[i] = 2.0f;
	}


	hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);

	int blockSize = 256;

	int numBlocks = (N + blockSize - 1) / blockSize;

	add<<<numBlocks, blockSize>>>(N, d_x, d_y);

	// cudaErrchk( cudaPeekAtLastError() );

	// cudaDeviceSynchronize();	
	hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);

	float maxError = 0.0f;
	for (int i = 0; i < N; i++) {
		maxError = fmax(maxError, fabs(h_y[i]-3.0f));
	}
	std::cout << "Max error: " << maxError << std::endl;

	hipFree(d_x);
	hipFree(d_y);
	free(h_x);
	free(h_y);
	return 0;
}
